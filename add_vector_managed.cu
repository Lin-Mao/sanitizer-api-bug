#include <hip/hip_runtime.h>
#include <iostream>

__global__ void vectorAdd(float *a, float *b, float *c, int n) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < n)
        c[i] = a[i] + b[i];
}

int main() {
    float *a, *b, *c;
    int n = 1 << 20;

    // Allocate managed memory
    hipMallocManaged(&a, n * sizeof(float));
    std::cout << "Managed memory allocated: a" << std::endl;

    hipMallocManaged(&b, n * sizeof(float));
    std::cout << "Managed memory allocated: b" << std::endl;

    hipMallocManaged(&c, n * sizeof(float));
    std::cout << "Managed memory allocated: c" << std::endl;

    // Initialize vectors
    for (int i = 0; i < n; i++) {
        a[i] = i;
        b[i] = i;
    }

    // Define grid and block dimensions
    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;

    // Launch kernel
    vectorAdd<<<gridSize, blockSize>>>(a, b, c, n);

    // Wait for GPU to finish
    hipDeviceSynchronize();

    // Check for errors
    hipError_t error = hipGetLastError();
    if(error != hipSuccess) {
        fprintf(stderr, "Error: %s\n", hipGetErrorString(error));
        return -1;
    }

    // Free memory
    hipFree(a);
    hipFree(b);
    hipFree(c);
    std::cout << "Done!" << std::endl;

    return 0;
}
